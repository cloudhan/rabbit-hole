#include "cpu/matmul.hpp"

#include <memory>
#include <type_traits>
#include "hipblas.h"

#define CUBLAS_CHECK(expr)                                                     \
  do {                                                                         \
    hipblasStatus_t err = (expr);                                               \
    if (err != HIPBLAS_STATUS_SUCCESS) {                                        \
      fprintf(stderr, "CUBLAS Error: %d at %s:%d\n", err, __FILE__, __LINE__); \
      exit(err);                                                               \
    }                                                                          \
  } while (0)

hipblasHandle_t get_cublas_handle() {
  using handle_deref_t = std::remove_pointer_t<hipblasHandle_t>;
  static std::shared_ptr<handle_deref_t> handle = []() {
    hipblasHandle_t tmp;
    CUBLAS_CHECK(hipblasCreate(&tmp));
    return std::shared_ptr<handle_deref_t>(tmp, [](hipblasHandle_t p) { CUBLAS_CHECK(hipblasDestroy(p)); });
  }();

  return handle.get();
}

namespace column_major {

MATMUL_SIGNATURE(matmul_reference) {
  float one = 1.0f;
  float zero = 0.0f;
  CUBLAS_CHECK(hipblasSgemm(
      get_cublas_handle(),
      HIPBLAS_OP_N, HIPBLAS_OP_N,
      m, n, k,
      &one,
      a, lda,
      b, ldb,
      &zero,
      c, ldc
  ));
}

}  // namespace column_major
