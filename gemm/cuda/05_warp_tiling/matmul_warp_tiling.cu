#include "hip/hip_runtime.h"
#include "cuda/matmul.cuh"

namespace column_major {

template <int M, int N>
using Acc = float[N][M];

template <int Size>
using Fragment = float[Size];

// Since we loop along k-axis for rank-1 update. Smem of a and b matrix should have the same K dimension
// template <int K, int Size> using Smem = float[K][Size];  // this cause ICE with nvcc 12.2
template <int K, int Size>
struct Array {
  float mem[K][Size];
};

// | ^ ^ ^  The transverse order of A and B.
// |/|/|/|  The A and B are assumed to be column majored.
// v v v |  The order ensure the access to global memory is coalesced.
// Threads cooperatively load from global memory to shared memory
// if number of elements in shared memory, then split the loading into multiple batches, along k-axis.
template <int NumThreads, int SmemShapeM, int SmemShapeK>
__device__ void store_smem_load_global_a(
    Array<SmemShapeK, SmemShapeM>& smem_a,
    int m,
    int k,
    const float* a,
    int lda,
    int a_basep
) {
  // Ensure the threads fill the column of A and the row of B. That is, when split only split along k-axis.
  // Otherwise, some elements will not be correctly handled.
  static_assert(NumThreads % SmemShapeM == 0);
  constexpr const auto SmemANumBatch = (SmemShapeM * SmemShapeK) / NumThreads;
  constexpr const auto SmemABatchShapeK = SmemShapeK / SmemANumBatch;

  const int A_i = SmemShapeM * blockIdx.x + threadIdx.x % SmemShapeM;
  const int A_batchp = a_basep + threadIdx.x / SmemShapeM;
#pragma unroll
  for (int smem_batch = 0; smem_batch < SmemShapeM * SmemShapeK / NumThreads; smem_batch++) {
    const auto smem_A_thread_i = threadIdx.x % SmemShapeM;
    const auto smem_A_thread_p = threadIdx.x / SmemShapeM + smem_batch * SmemABatchShapeK;
    const auto A_p = A_batchp + smem_batch * SmemABatchShapeK;
    smem_a.mem[smem_A_thread_p][smem_A_thread_i] = A_i >= m || A_p >= k ? 0 : a[A_i * 1 + A_p * lda];
  }
}

template <int NumThreads, int SmemShapeK, int SmemShapeN>
__device__ void store_smem_load_global_b(
    Array<SmemShapeK, SmemShapeN>& smem_b,
    int k,
    int n,
    const float* b,
    int ldb,
    int b_basep
) {
  static_assert(NumThreads % SmemShapeN == 0);
  constexpr const auto SmemBNumBatch = (SmemShapeN * SmemShapeK) / NumThreads;
  constexpr const auto SmemBBatchShapeK = SmemShapeK / SmemBNumBatch;

  const int B_batchp = b_basep + threadIdx.x % SmemBBatchShapeK;
  const int B_j = SmemShapeN * blockIdx.y + threadIdx.x / SmemBBatchShapeK;
#pragma unroll
  for (int smem_batch = 0; smem_batch < SmemShapeN * SmemShapeK / NumThreads; smem_batch++) {
    const auto smem_B_thread_p = threadIdx.x % SmemBBatchShapeK + smem_batch * SmemBBatchShapeK;
    const auto smem_B_thread_j = threadIdx.x / SmemBBatchShapeK;
    const auto B_p = B_batchp + smem_batch * SmemBBatchShapeK;
    smem_b.mem[smem_B_thread_p][smem_B_thread_j] = B_p >= k || B_j >= n ? 0 : b[B_p * 1 + B_j * ldb];
  }
}

template <int FragmentSize, int SmemShapeK, int SmemShapeM /*or SmemShapeN*/>
__device__ void load_fragment(
    Fragment<FragmentSize>& frag_a,               // or frag_b
    const Array<SmemShapeK, SmemShapeM>& smem_a,  // or smem_b
    int smem_a_thread_p,                          // or smem_b_thread_p
    int smem_a_thread_i                           // or smem_b_thread_j
) {
  static_assert(SmemShapeM % FragmentSize == 0);
  auto ptr = &smem_a.mem[smem_a_thread_p][smem_a_thread_i];
#pragma unroll
  for (int f = 0; f < FragmentSize; f++, ptr += 1) {
    frag_a[f] = *ptr;
  }
}

template <int ThreadShapeM, int ThreadShapeN>
__device__ void rank1_update(Acc<ThreadShapeM, ThreadShapeN>& acc, const Fragment<ThreadShapeM>& frag_a, const Fragment<ThreadShapeN>& frag_b) {
  // rank-1 update to acc registers
#pragma unroll
  for (int j = 0; j < ThreadShapeN; j++) {
#pragma unroll
    for (int i = 0; i < ThreadShapeM; i++) {
      acc[j][i] += frag_a[i] * frag_b[j];
    }
  }
}

template <int ThreadShapeM, int ThreadShapeN>
__device__ void acc_store(int m, int n, float* C, int ldc, Acc<ThreadShapeM, ThreadShapeN>& acc, int thread_i, int thread_j) {
  // store acc registers results to C
  float* thread_c = &C[thread_i * 1 + thread_j * ldc];
#pragma unroll
  for (int b = 0; b < ThreadShapeN; b++) {
    if (thread_j + b < n) {
#pragma unroll
      for (int a = 0; a < ThreadShapeM; a++) {
        if (thread_i + a < m) {
          thread_c[a * 1 + b * ldc] = acc[b][a];
        }
      }
    }
  }
}

// Laucnh a 1D CTA(threadblock)
// Each CTA process CtaShapeM x CtaShapeN tile of C
// CTA load SmemShapeM x SmemShapeK and SmemShapeK x SmemShapeN of A and B from global memory to shared memory
// CTA contains NumThreads/32 of warps. Each warp is then organized to process WarpShapeM x WarpShapeN of data of C,
//     threads in a warp are layouted as column major and warps in a CTA are layouted in column major.
// Each thread process ThreadShapeM x ThreadShapeN of **collocated** data
// Each thread then load (ThreadShapeM + ThreadShapeN) of elements, and do ThreadShapeM * ThreadShapeN of FMAs.
template <int NumThreads, int CtaShapeM, int CtaShapeN, int SmemShapeK, int WarpShapeM, int WarpShapeN, int ThreadShapeM, int ThreadShapeN>
MATMUL_KERNEL_SIGNATURE(matmul_kernel_warp_tiling) {
  constexpr const auto SmemShapeM = CtaShapeM;
  constexpr const auto SmemShapeN = CtaShapeN;
  static_assert((SmemShapeM * SmemShapeK) % NumThreads == 0 && (SmemShapeN * SmemShapeK) % NumThreads == 0);
  static_assert((CtaShapeM * CtaShapeN / (ThreadShapeM * ThreadShapeN)) == NumThreads);
  // static_assert((WarpShapeM * WarpShapeN / (ThreadShapeM * ThreadShapeN)) == warpSize);
  static_assert(CtaShapeM % WarpShapeM == 0 && CtaShapeN % WarpShapeN == 0);
  static_assert(WarpShapeM % ThreadShapeM == 0 && WarpShapeN % ThreadShapeN == 0);

  __shared__ Array<SmemShapeK, SmemShapeM> smem_a;
  __shared__ Array<SmemShapeK, SmemShapeN> smem_b;

  const int warp_id = threadIdx.x / warpSize;  // can be accessed with special registers %warpid in PTX
  const int lane_id = threadIdx.x % warpSize;  // can be accessed with special registers %laneid in PTX
  // from cta to warp
  const int cta_warp_i = (warp_id % (CtaShapeM / WarpShapeM)) * WarpShapeM;
  const int cta_warp_j = (warp_id / (CtaShapeM / WarpShapeM)) * WarpShapeN;
  // from cta to thread
  const int cta_thread_i = cta_warp_i + (lane_id % (WarpShapeM / ThreadShapeM)) * ThreadShapeM;
  const int cta_thread_j = cta_warp_j + (lane_id / (WarpShapeM / ThreadShapeM)) * ThreadShapeN;

  Acc<ThreadShapeM, ThreadShapeN> acc{};
  Fragment<ThreadShapeM> frag_a;
  Fragment<ThreadShapeN> frag_b;

  for (int p = 0; p < k; p += SmemShapeK) {
    // load A, B to smem_A, smem_B
    store_smem_load_global_a<NumThreads, SmemShapeM, SmemShapeK>(smem_a, m, k, a, lda, p);
    store_smem_load_global_b<NumThreads, SmemShapeK, SmemShapeN>(smem_b, k, n, b, ldb, p);
    __syncthreads();

    // each thread then load from shared memory to register and perform the rank-1 update
    // threads are not organized naively as previous kernel, instead, each warp now have a shape.
    const auto smem_A_thread_i = cta_thread_i;
    const auto smem_B_thread_j = cta_thread_j;
    // #pragma unroll
    for (int smem_AB_thread_p = 0; smem_AB_thread_p < SmemShapeK; smem_AB_thread_p++) {
      // register load
      load_fragment<ThreadShapeM, SmemShapeK, SmemShapeM>(frag_a, smem_a, smem_AB_thread_p, smem_A_thread_i);
      load_fragment<ThreadShapeN, SmemShapeK, SmemShapeN>(frag_b, smem_b, smem_AB_thread_p, smem_B_thread_j);

      rank1_update<ThreadShapeM, ThreadShapeN>(acc, frag_a, frag_b);
    }
    __syncthreads();
  }

  // store acc registers results to C
  const int cta_i = CtaShapeM * blockIdx.x;
  const int cta_j = CtaShapeN * blockIdx.y;
  const int thread_i = cta_i + cta_thread_i;
  const int thread_j = cta_j + cta_thread_j;
  acc_store(m, n, c, ldc, acc, thread_i, thread_j);
}

#define MATMUL_KERNEL_LAUNCH(name, num_threads, cta_shape_m, cta_shape_n, smem_shape_k, warp_shape_m, warp_shape_n, thread_shape_m, thread_shape_n)                                            \
  MATMUL_SIGNATURE(launch_##name##_##num_threads##t_cta##cta_shape_m##x##cta_shape_n##_smem##smem_shape_k##_warp##warp_shape_m##x##warp_shape_n##_thread##thread_shape_m##x##thread_shape_n) { \
    dim3 threads(num_threads);                                                                                                                                                                 \
    dim3 blocks(ceil_div<int64_t>(m, cta_shape_m), ceil_div<int64_t>(n, cta_shape_n));                                                                                                         \
    name<num_threads, cta_shape_m, cta_shape_n, smem_shape_k, warp_shape_m, warp_shape_n, thread_shape_m, thread_shape_n><<<                                                                   \
        blocks, threads, 0, nullptr>>>(m, n, k, a, lda, b, ldb, c, ldc);                                                                                                                       \
    CUDA_CHECK(hipGetLastError());                                                                                                                                                            \
  }

MATMUL_KERNEL_LAUNCH(matmul_kernel_warp_tiling, 256, 128, 128, 8, 128, 16, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_warp_tiling, 256, 128, 128, 8, 64, 32, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_warp_tiling, 256, 128, 128, 8, 32, 64, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_warp_tiling, 256, 128, 128, 8, 16, 128, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_warp_tiling, 256, 128, 128, 16, 128, 16, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_warp_tiling, 256, 128, 128, 16, 64, 32, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_warp_tiling, 256, 128, 128, 16, 32, 64, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_warp_tiling, 256, 128, 128, 16, 16, 128, 8, 8);

MATMUL_DMODULE(m) {
  REGISTER(launch_matmul_kernel_warp_tiling_256t_cta128x128_smem8_warp128x16_thread8x8);
  REGISTER(launch_matmul_kernel_warp_tiling_256t_cta128x128_smem8_warp64x32_thread8x8);
  REGISTER(launch_matmul_kernel_warp_tiling_256t_cta128x128_smem8_warp32x64_thread8x8);
  REGISTER(launch_matmul_kernel_warp_tiling_256t_cta128x128_smem8_warp16x128_thread8x8);
  REGISTER(launch_matmul_kernel_warp_tiling_256t_cta128x128_smem16_warp128x16_thread8x8);
  REGISTER(launch_matmul_kernel_warp_tiling_256t_cta128x128_smem16_warp64x32_thread8x8);
  REGISTER(launch_matmul_kernel_warp_tiling_256t_cta128x128_smem16_warp32x64_thread8x8);
  REGISTER(launch_matmul_kernel_warp_tiling_256t_cta128x128_smem16_warp16x128_thread8x8);
}

}  // namespace column_major
