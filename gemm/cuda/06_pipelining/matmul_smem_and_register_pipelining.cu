#include "hip/hip_runtime.h"
#include "cuda/matmul.cuh"

namespace column_major {

template <int M, int N>
using Acc = float[N][M];

template <int Size>
struct Registers {
  float reg[Size];
};

template <int Size>
using Fragment = float[Size];

// Since we loop along k-axis for rank-1 update. Smem of a and b matrix should have the same K dimension
// template <int K, int Size> using Smem = float[K][Size];  // this cause ICE with nvcc 12.2
template <int K, int Size>
struct Array {
  float mem[K][Size];
};

// | ^ ^ ^  The transverse order of A and B.
// |/|/|/|  The A and B are assumed to be column majored.
// v v v |  The order ensure the access to global memory is coalesced.
// Threads cooperatively load from global memory to shared memory
// if number of elements in shared memory, then split the loading into multiple batches, along k-axis.
template <int NumThreads, int SmemShapeM, int SmemShapeK, int SmemANumBatch = (SmemShapeM * SmemShapeK) / NumThreads>
__device__ void load_global_a(
    Registers<SmemANumBatch>& reg_a,
    int m,
    int k,
    const float* a,
    int lda,
    int a_thread_p
) {
  // Ensure the threads fill the column of A and the row of B. That is, when split only split along k-axis.
  // Otherwise, some elements will not be correctly handled.
  static_assert(NumThreads % SmemShapeM == 0);
  constexpr const auto SmemABatchShapeK = SmemShapeK / SmemANumBatch;

  const int A_i = SmemShapeM * blockIdx.x + threadIdx.x % SmemShapeM;
  const int A_batchp = a_thread_p + threadIdx.x / SmemShapeM;
#pragma unroll
  for (int batch = 0; batch < SmemANumBatch; batch++) {
    const auto A_p = A_batchp + batch * SmemABatchShapeK;
    reg_a.reg[batch] = A_i >= m || A_p >= k ? 0 : a[A_i * 1 + A_p * lda];
  }
}

template <int NumThreads, int SmemShapeK, int SmemShapeN, int SmemBNumBatch = (SmemShapeN * SmemShapeK) / NumThreads>
__device__ void load_global_b(
    Registers<SmemBNumBatch>& reg_b,
    int k,
    int n,
    const float* b,
    int ldb,
    int b_thread_p
) {
  static_assert(NumThreads % SmemShapeN == 0);
  constexpr const auto SmemBBatchShapeK = SmemShapeK / SmemBNumBatch;

  const int B_batchp = b_thread_p + threadIdx.x % SmemBBatchShapeK;
  const int B_j = SmemShapeN * blockIdx.y + threadIdx.x / SmemBBatchShapeK;
#pragma unroll
  for (int batch = 0; batch < SmemShapeN * SmemShapeK / NumThreads; batch++) {
    const auto B_p = B_batchp + batch * SmemBBatchShapeK;
    reg_b.reg[batch] = B_p >= k || B_j >= n ? 0 : b[B_p * 1 + B_j * ldb];
  }
}

template <int NumThreads, int SmemShapeM, int SmemShapeK, int SmemANumBatch = (SmemShapeM * SmemShapeK) / NumThreads>
__device__ void store_smem_a(
    Array<SmemShapeK, SmemShapeM>& smem_a,
    const Registers<SmemANumBatch>& reg_a
) {
  static_assert(NumThreads % SmemShapeM == 0);
  constexpr const auto SmemABatchShapeK = SmemShapeK / SmemANumBatch;
#pragma unroll
  for (int batch = 0; batch < SmemANumBatch; batch++) {
    const auto smem_A_thread_i = threadIdx.x % SmemShapeM;
    const auto smem_A_thread_p = threadIdx.x / SmemShapeM + batch * SmemABatchShapeK;
    smem_a.mem[smem_A_thread_p][smem_A_thread_i] = reg_a.reg[batch];
  }
}

template <int NumThreads, int SmemShapeK, int SmemShapeN, int SmemBNumBatch = (SmemShapeN * SmemShapeK) / NumThreads>
__device__ void store_smem_b(
    Array<SmemShapeK, SmemShapeN>& smem_b,
    const Registers<SmemBNumBatch>& reg_b
) {
  static_assert(NumThreads % SmemShapeN == 0);
  constexpr const auto SmemBBatchShapeK = SmemShapeK / SmemBNumBatch;
#pragma unroll
  for (int batch = 0; batch < SmemShapeN * SmemShapeK / NumThreads; batch++) {
    const auto smem_B_thread_p = threadIdx.x % SmemBBatchShapeK + batch * SmemBBatchShapeK;
    const auto smem_B_thread_j = threadIdx.x / SmemBBatchShapeK;
    smem_b.mem[smem_B_thread_p][smem_B_thread_j] = reg_b.reg[batch];
  }
}

template <int FragmentSize, int SmemShapeK, int SmemShapeM /*or SmemShapeN*/>
__device__ void load_fragment(
    Fragment<FragmentSize>& frag_a,               // or frag_b
    const Array<SmemShapeK, SmemShapeM>& smem_a,  // or smem_b
    int smem_a_thread_p,                          // or smem_b_thread_p
    int smem_a_thread_i                           // or smem_b_thread_j
) {
  static_assert(SmemShapeM % FragmentSize == 0);
  auto ptr = &smem_a.mem[smem_a_thread_p][smem_a_thread_i];
#pragma unroll
  for (int f = 0; f < FragmentSize; f++, ptr += 1) {
    frag_a[f] = *ptr;
  }
}

template <int ThreadShapeM, int ThreadShapeN>
__device__ void rank1_update(Acc<ThreadShapeM, ThreadShapeN>& acc, const Fragment<ThreadShapeM>& frag_a, const Fragment<ThreadShapeN>& frag_b) {
  // rank-1 update to acc registers
#pragma unroll
  for (int j = 0; j < ThreadShapeN; j++) {
#pragma unroll
    for (int i = 0; i < ThreadShapeM; i++) {
      acc[j][i] += frag_a[i] * frag_b[j];
    }
  }
}

template <int ThreadShapeM, int ThreadShapeN>
__device__ void acc_store(int m, int n, float* C, int ldc, Acc<ThreadShapeM, ThreadShapeN>& acc, int thread_i, int thread_j) {
  // store acc registers results to C
  float* thread_c = &C[thread_i * 1 + thread_j * ldc];
#pragma unroll
  for (int b = 0; b < ThreadShapeN; b++) {
    if (thread_j + b < n) {
#pragma unroll
      for (int a = 0; a < ThreadShapeM; a++) {
        if (thread_i + a < m) {
          thread_c[a * 1 + b * ldc] = acc[b][a];
        }
      }
    }
  }
}

// Laucnh a 1D CTA(threadblock)
// Each CTA process CtaShapeM x CtaShapeN tile of C
// Preload a block of shared memory
// In the loop
// CTA load SmemShapeM x SmemShapeK and SmemShapeK x SmemShapeN of A and B from global memory to shared memory
// Each thread process ThreadShapeM x ThreadShapeN of **collocated** data
// Each thread then load (ThreadShapeM + ThreadShapeN) of elements, and do ThreadShapeM * ThreadShapeN of FMAs.
template <int NumThreads, int CtaShapeM, int CtaShapeN, int SmemShapeK, int ThreadShapeM, int ThreadShapeN>
MATMUL_KERNEL_SIGNATURE(matmul_kernel_pipelining) {
  constexpr const auto SmemShapeM = CtaShapeM;
  constexpr const auto SmemShapeN = CtaShapeN;
  static_assert((SmemShapeM * SmemShapeK) % NumThreads == 0 && (SmemShapeN * SmemShapeK) % NumThreads == 0);
  static_assert((CtaShapeM * CtaShapeN / (ThreadShapeM * ThreadShapeN)) == NumThreads);
  static_assert(CtaShapeM % ThreadShapeM == 0 && CtaShapeN % ThreadShapeN == 0);

  __shared__ Array<SmemShapeK, SmemShapeM> smem_a[2];
  __shared__ Array<SmemShapeK, SmemShapeN> smem_b[2];
  Registers<SmemShapeM * SmemShapeK / NumThreads> staging_a;
  Registers<SmemShapeN * SmemShapeK / NumThreads> staging_b;

  const int cta_i = CtaShapeM * blockIdx.x;
  const int cta_j = CtaShapeN * blockIdx.y;
  const int thread_i = cta_i + (threadIdx.x % (CtaShapeM / ThreadShapeM)) * ThreadShapeM;
  const int thread_j = cta_j + (threadIdx.x / (CtaShapeM / ThreadShapeM)) * ThreadShapeN;

  Acc<ThreadShapeM, ThreadShapeN> acc{};
  Fragment<ThreadShapeM> frag_a[2];
  Fragment<ThreadShapeN> frag_b[2];

  // pre-load first block of A, B to smem_A, smem_B
  load_global_a<NumThreads, SmemShapeM, SmemShapeK>(staging_a, m, k, a, lda, 0);
  load_global_b<NumThreads, SmemShapeK, SmemShapeN>(staging_b, k, n, b, ldb, 0);
  for (int p = 0; p < k; p += SmemShapeK) {
    store_smem_a<NumThreads, SmemShapeM, SmemShapeK>(smem_a[(p / SmemShapeK) % 2], staging_a);
    store_smem_b<NumThreads, SmemShapeK, SmemShapeN>(smem_b[(p / SmemShapeK) % 2], staging_b);
    __syncthreads();
    if (p + SmemShapeK < k) {
      load_global_a<NumThreads, SmemShapeM, SmemShapeK>(staging_a, m, k, a, lda, p + SmemShapeK);
      load_global_b<NumThreads, SmemShapeK, SmemShapeN>(staging_b, k, n, b, ldb, p + SmemShapeK);
    }

    // each thread then load from shared memory to register and perform the rank-1 update
    const auto smem_A_thread_i = threadIdx.x % (CtaShapeM / ThreadShapeM) * ThreadShapeM;
    const auto smem_B_thread_j = threadIdx.x / (CtaShapeM / ThreadShapeM) * ThreadShapeN;

    // pre-load first fragment of registers
    load_fragment<ThreadShapeM, SmemShapeK, SmemShapeM>(frag_a[0], smem_a[(p / SmemShapeK) % 2], 0, smem_A_thread_i);
    load_fragment<ThreadShapeN, SmemShapeK, SmemShapeN>(frag_b[0], smem_b[(p / SmemShapeK) % 2], 0, smem_B_thread_j);
#pragma unroll
    for (int smem_AB_thread_p = 0; smem_AB_thread_p < SmemShapeK; smem_AB_thread_p++) {
      // pre-load next fragment of registers
      if (smem_AB_thread_p + 1 < SmemShapeK) {
        load_fragment<ThreadShapeM, SmemShapeK, SmemShapeM>(frag_a[(smem_AB_thread_p + 1) % 2], smem_a[(p / SmemShapeK) % 2], smem_AB_thread_p + 1, smem_A_thread_i);
        load_fragment<ThreadShapeN, SmemShapeK, SmemShapeN>(frag_b[(smem_AB_thread_p + 1) % 2], smem_b[(p / SmemShapeK) % 2], smem_AB_thread_p + 1, smem_B_thread_j);
      }

      rank1_update<ThreadShapeM, ThreadShapeN>(acc, frag_a[smem_AB_thread_p % 2], frag_b[smem_AB_thread_p % 2]);
    }
  }

  // store acc registers results to C
  acc_store(m, n, c, ldc, acc, thread_i, thread_j);
}

#define MATMUL_KERNEL_LAUNCH(name, num_threads, cta_shape_m, cta_shape_n, smem_shape_k, thread_shape_m, thread_shape_n)                                  \
  MATMUL_SIGNATURE(launch_##name##_##num_threads##t_cta##cta_shape_m##x##cta_shape_n##_smem##smem_shape_k##_thread##thread_shape_m##x##thread_shape_n) { \
    dim3 threads(num_threads);                                                                                                                           \
    dim3 blocks(ceil_div<int64_t>(m, cta_shape_m), ceil_div<int64_t>(n, cta_shape_n));                                                                   \
    name<num_threads, cta_shape_m, cta_shape_n, smem_shape_k, thread_shape_m, thread_shape_n><<<                                                         \
        blocks, threads, 0, nullptr>>>(m, n, k, a, lda, b, ldb, c, ldc);                                                                                 \
    CUDA_CHECK(hipGetLastError());                                                                                                                      \
  }

MATMUL_KERNEL_LAUNCH(matmul_kernel_pipelining, 256, 128, 128, 4, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_pipelining, 256, 128, 128, 8, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_pipelining, 256, 128, 128, 16, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_pipelining, 256, 128, 128, 24, 8, 8);

MATMUL_DMODULE(m) {
  REGISTER(launch_matmul_kernel_pipelining_256t_cta128x128_smem4_thread8x8);
  REGISTER(launch_matmul_kernel_pipelining_256t_cta128x128_smem8_thread8x8);
  REGISTER(launch_matmul_kernel_pipelining_256t_cta128x128_smem16_thread8x8);
  REGISTER(launch_matmul_kernel_pipelining_256t_cta128x128_smem24_thread8x8);
}

}  // namespace column_major
