#include "hip/hip_runtime.h"
#include <cute/layout.hpp>                     // make_shape, make_stride, make_layout
#include <cute/pointer.hpp>                    // make_gmem_ptr
#include <cute/tensor.hpp>                     // make_tensor
#include <cute/numeric/integral_constant.hpp>  // _1

#include "cuda/matmul.cuh"

using namespace cute;

// not going to implement copy/store with multiple batches in this kernel.
template <int NumThreads, int SmemShapeM, int SmemShapeK, int VecSize, typename RegT, typename GmemT, typename CoordT>
__device__ void load_global_a(RegT& rA, const GmemT& gA, const CoordT& cA, int m, int k) {
  static_assert(is_rmem<typename RegT::engine_type>());
  static_assert(VecSize == 1 || VecSize == 2 || VecSize == 4);
  constexpr const auto ThrVal = make_layout(make_layout(Int<NumThreads>{}, Int<VecSize>{}), make_layout(Int<VecSize>{}));
  const auto ld_gA = gA.compose(ThrVal)(threadIdx.x, _);
  const auto ld_cA = cA.compose(ThrVal);
  clear(rA);
#pragma unroll
  for (int i = 0; i < VecSize; i++) {
    if (elem_less(ld_cA(threadIdx.x, i), make_coord(m, k))) {
      rA(i) = ld_gA(i);
    }
  }
}

template <int NumThreads, int SmemShapeN, int SmemShapeK, int VecSize, typename RegT, typename GmemT, typename CoordT>
__device__ void load_global_b(RegT& rB, const GmemT& gB, const CoordT& cB, int n, int k) {
  static_assert(is_rmem<typename RegT::engine_type>());
  static_assert(VecSize == 1 || VecSize == 2 || VecSize == 4);
  constexpr const auto NumLoadK = SmemShapeK / VecSize; // number of load along k
  static_assert(NumLoadK > 0);
  constexpr const auto ThrVal = make_layout(
      make_layout(make_shape(Int<NumThreads/NumLoadK>{}, Int<NumLoadK>{}), make_stride(Int<SmemShapeK>{}, Int<VecSize>{})),
      make_layout(Int<VecSize>{})
      );
  const auto ld_gB = gB.compose(ThrVal)(threadIdx.x, _);
  const auto ld_cB = cB.compose(ThrVal);
  clear(rB);
#pragma unroll
  for (int i = 0; i < VecSize; i++) {
    if (elem_less(ld_cB(threadIdx.x, i), make_coord(n, k))) {
      rB(i) = ld_gB(i);
    }
  }
}

template <int NumThreads, int SmemShapeM, int SmemShapeK, int VecSize, typename SmemT, typename RegT>
__device__ void store_smem_a(SmemT& sA, const RegT& rA, int m, int k) {
  static_assert(is_rmem<typename RegT::engine_type>());
  static_assert(VecSize == 1 || VecSize == 2 || VecSize == 4);
  constexpr const auto ThrVal = make_layout(make_layout(Int<NumThreads>{}, Int<VecSize>{}), make_layout(Int<VecSize>{}));
  auto st_sA = sA.compose(ThrVal)(threadIdx.x, _);
  copy(rA, st_sA);
}

template <int NumThreads, int SmemShapeN, int SmemShapeK, int VecSize, typename SmemT, typename RegT>
__device__ void store_smem_b(SmemT& sB, const RegT& rB, int n, int k) {
  static_assert(is_rmem<typename RegT::engine_type>());
  static_assert(VecSize == 1 || VecSize == 2 || VecSize == 4);
  constexpr const auto NumLoadK = SmemShapeK / VecSize; // number of load along k
  static_assert(NumLoadK > 0);
  constexpr const auto ThrVal = make_layout(
      make_layout(make_shape(Int<NumThreads/NumLoadK>{}, Int<NumLoadK>{}), make_stride(Int<SmemShapeK>{}, Int<VecSize>{})),
      make_layout(Int<VecSize>{})
      );
  auto st_sB = sB.compose(ThrVal)(threadIdx.x, _);
  copy(rB, st_sB);
}

template <typename AccT, typename FragAT, typename FragBT>
__device__ void rank1_update(const FragAT& fragA, const FragBT& fragB, AccT& acc) {
  static_assert(is_rmem<typename AccT::engine_type>() && is_rmem<typename FragAT::engine_type>() && is_rmem<typename FragBT::engine_type>());
  constexpr const auto ThreadShapeM = size<0>(typename AccT::layout_type{});
  constexpr const auto ThreadShapeN = size<1>(typename AccT::layout_type{});
#pragma unroll
  for (int j = 0; j < ThreadShapeN; j++) {
#pragma unroll
    for (int i = 0; i < ThreadShapeM; i++) {
      acc(i, j) += fragA(i) * fragB(j);
    }
  }
}

template <typename AccT, typename CtaCT, typename CoordT>
__device__ void acc_store(const AccT& acc, CtaCT& threadC, CoordT thread_cC, int m, int n) {
  if (elem_less(thread_cC(size<0>(thread_cC) - 1, size<1>(thread_cC) - 1), make_coord(m, n))) {  // fast path
    copy(acc, threadC);
  } else {
    const auto [thread_i, thread_j] = thread_cC(_0{}, _0{});
    auto predA = make_tensor<bool>(Int<size<0>(thread_cC)>{});
    for (int i = 0; i < size<0>(predA); i++) {
      predA(i) = thread_i + i < m;
    }
    constexpr const auto ThreadShapeN = size<1>(typename AccT::layout_type{});
#pragma unroll
    for (int j = 0; j < ThreadShapeN; j++) {
      if (thread_j + j < n) {
        copy_if(predA, acc(_, j), threadC(_, j));
      }
    }
  }
}

namespace column_major {
template <int NumThreads, int CtaShapeM, int CtaShapeN, int SmemShapeK, int ThreadShapeM, int ThreadShapeN>
__launch_bounds__(NumThreads, 2)
MATMUL_KERNEL_SIGNATURE(matmul_smem_and_register_pipelining_2) {
  constexpr const auto SmemShapeM = CtaShapeM;
  constexpr const auto SmemShapeN = CtaShapeN;

  // original matrix
  const auto mA = make_tensor(make_gmem_ptr(a), make_layout(make_shape(m, k), make_stride(_1{}, lda)));  // col-major, indexed as (m, k)
  const auto mB = make_tensor(make_gmem_ptr(b), make_layout(make_shape(n, k), make_stride(ldb, _1{})));  // col-major storage, row-major indexing, indexed as (n, k), effectively a "transposed view"
  auto mC = make_tensor(make_gmem_ptr(c), make_layout(make_shape(m, n), make_stride(_1{}, ldc)));        // col-major, indexed as (m, n)

  // coordinate matrix
  const auto cA = make_identity_tensor(make_shape(m, k));
  const auto cB = make_identity_tensor(make_shape(n, k));
  const auto cC = make_identity_tensor(make_shape(m, n));

  const auto CtaShape = make_shape(Int<SmemShapeM>{}, Int<SmemShapeN>{}, Int<SmemShapeK>{});
  const auto cta_coord = make_coord(blockIdx.x, blockIdx.y, _);

  // a local view (in CuTe term, local tile) this CTA will need to process
  const auto ctaA = local_tile(mA, CtaShape, cta_coord, make_step(_1{}, _, _1{}));
  const auto ctaB = local_tile(mB, CtaShape, cta_coord, make_step(_, _1{}, _1{}));
  auto ctaC = local_tile(mC, CtaShape, cta_coord, make_step(_1{}, _1{}, _));

  const auto cta_cA = local_tile(cA, CtaShape, cta_coord, make_step(_1{}, _, _1{}));
  const auto cta_cB = local_tile(cB, CtaShape, cta_coord, make_step(_, _1{}, _1{}));
  const auto cta_cC = local_tile(cC, CtaShape, cta_coord, make_step(_1{}, _1{}, _));

  constexpr const int SmemAPad = 0;
  constexpr const int SmemBPad = 4;
  constexpr const auto SmemALayout = make_layout(make_shape(Int<SmemShapeM>{}, Int<SmemShapeK>{}), make_stride(_1{}, Int<SmemShapeM + SmemAPad>{}));
  constexpr const auto SmemBLayout = make_layout(make_shape(Int<SmemShapeN>{}, Int<SmemShapeK>{}), make_stride(_1{}, Int<SmemShapeN + SmemBPad>{}));
  // double buffering
  __shared__ float smemA[cosize(SmemALayout)];
  __shared__ float smemB[cosize(SmemBLayout)];
  auto sA = make_tensor(make_smem_ptr(smemA), SmemALayout);  // (m, k)
  auto sB = make_tensor(make_smem_ptr(smemB), SmemBLayout);  // (n, k)

  const auto stripe_gA = local_tile(ctaA, make_tile(Int<SmemShapeM>{}, Int<SmemShapeK>{}), make_coord(blockIdx.x, _));  // A(blockIdx.x*SmemShapeM:(blockIdx.x+1)*SmemShapeM, :)
  const auto stripe_gB = local_tile(ctaB, make_tile(Int<SmemShapeN>{}, Int<SmemShapeK>{}), make_coord(blockIdx.y, _));  // B(blockIdx.y*SmemShapeN:(blockIdx.y+1)*SmemShapeN, :)

  const auto stripe_cA = local_tile(cta_cA, make_tile(Int<SmemShapeM>{}, Int<SmemShapeK>{}), make_coord(blockIdx.x, _));
  const auto stripe_cB = local_tile(cta_cB, make_tile(Int<SmemShapeN>{}, Int<SmemShapeK>{}), make_coord(blockIdx.y, _));

  using fragA_t = decltype(make_fragment_like<float>(Int<ThreadShapeM>{}));
  using fragB_t = decltype(make_fragment_like<float>(Int<ThreadShapeN>{}));
  fragA_t fragA[2] = {make_fragment_like<float>(Int<ThreadShapeM>{}), make_fragment_like<float>(Int<ThreadShapeM>{})};
  fragB_t fragB[2] = {make_fragment_like<float>(Int<ThreadShapeN>{}), make_fragment_like<float>(Int<ThreadShapeN>{})};
  auto acc = make_fragment_like<float>(make_shape(Int<ThreadShapeM>{}, Int<ThreadShapeN>{}));
  clear(acc);

  constexpr const auto SmemALoadStoreVec = (SmemShapeM * SmemShapeK) / NumThreads;  // VecSize
  constexpr const auto SmemBLoadStoreVec = (SmemShapeN * SmemShapeK) / NumThreads;  // VecSize
  auto staging_a = make_fragment_like<float>(Int<SmemALoadStoreVec>{});
  auto staging_b = make_fragment_like<float>(Int<SmemBLoadStoreVec>{});

  int p_tile_count = (k - 1) / SmemShapeK + 1;
  int p_tile_next = 0;

  load_global_a<NumThreads, SmemShapeM, SmemShapeK, SmemALoadStoreVec>(staging_a, stripe_gA(_, _, _0{}, p_tile_next), stripe_cA(_, _, _0{}, p_tile_next), m, k);
  load_global_b<NumThreads, SmemShapeN, SmemShapeK, SmemBLoadStoreVec>(staging_b, stripe_gB(_, _, _0{}, p_tile_next), stripe_cB(_, _, _0{}, p_tile_next), n, k);
  if (--p_tile_count > 0) {
    ++p_tile_next;
  }
  store_smem_a<NumThreads, SmemShapeM, SmemShapeK, SmemALoadStoreVec>(sA, staging_a, m, k);
  store_smem_b<NumThreads, SmemShapeN, SmemShapeK, SmemBLoadStoreVec>(sB, staging_b, n, k);
  __syncthreads();

  const auto stripe_sA = local_tile(sA, make_tile(Int<ThreadShapeM>{}, Int<SmemShapeK>{}), threadIdx.x % (CtaShapeM / ThreadShapeM));
  const auto stripe_sB = local_tile(sB, make_tile(Int<ThreadShapeN>{}, Int<SmemShapeK>{}), threadIdx.x / (CtaShapeM / ThreadShapeM));

  copy(stripe_sA(_, 0, _0{}), fragA[0]);  // load_fragment a
  copy(stripe_sB(_, 0, _0{}), fragB[0]);  // load_fragment b

  const auto num_smem_block = size<3>(stripe_gA);
#pragma unroll 1  // no unroll
  for (int block_p = 0; block_p < num_smem_block; block_p++) {
#pragma unroll
    for (int smem_AB_thread_p = 0; smem_AB_thread_p < SmemShapeK; smem_AB_thread_p++) {
      if (smem_AB_thread_p == SmemShapeK - 1) {
        __syncthreads();
        store_smem_a<NumThreads, SmemShapeM, SmemShapeK, SmemALoadStoreVec>(sA, staging_a, m, k);
        store_smem_b<NumThreads, SmemShapeN, SmemShapeK, SmemBLoadStoreVec>(sB, staging_b, n, k);
        __syncthreads();
        if (--p_tile_count > 0) {
          ++p_tile_next;
        }
      }

      copy(stripe_sA(_, (smem_AB_thread_p + 1) % SmemShapeK, _0{}), fragA[(smem_AB_thread_p + 1) % 2]);  // load_fragment a
      copy(stripe_sB(_, (smem_AB_thread_p + 1) % SmemShapeK, _0{}), fragB[(smem_AB_thread_p + 1) % 2]);  // load_fragment b

      if (smem_AB_thread_p == 0) {
        load_global_a<NumThreads, SmemShapeM, SmemShapeK, SmemALoadStoreVec>(staging_a, stripe_gA(_, _, _0{}, p_tile_next), stripe_cA(_, _, _0{}, p_tile_next), m, k);
        load_global_b<NumThreads, SmemShapeN, SmemShapeK, SmemBLoadStoreVec>(staging_b, stripe_gB(_, _, _0{}, p_tile_next), stripe_cB(_, _, _0{}, p_tile_next), n, k);
      }

      rank1_update(fragA[smem_AB_thread_p % 2], fragB[smem_AB_thread_p % 2], acc);
    }
  }

  constexpr const auto MapToThread = make_layout(make_shape(Int<CtaShapeM / ThreadShapeM>{}, Int<CtaShapeN / ThreadShapeN>{}));
  auto [cta_thread_i, cta_thread_j] = idx2crd(MapToThread(threadIdx.x), MapToThread.shape(), MapToThread.stride());
  cta_thread_i *= ThreadShapeM;
  cta_thread_j *= ThreadShapeN;

  auto threadC = local_tile(ctaC, make_tile(Int<ThreadShapeM>{}, Int<ThreadShapeN>{}), make_coord(cta_thread_i / ThreadShapeM, cta_thread_j / ThreadShapeN));
  auto thread_cC = local_tile(cta_cC, make_tile(Int<ThreadShapeM>{}, Int<ThreadShapeN>{}), make_coord(cta_thread_i / ThreadShapeM, cta_thread_j / ThreadShapeN));
  acc_store(acc, threadC, thread_cC, m, n);
}

#define MATMUL_KERNEL_LAUNCH(name, num_threads, cta_shape_m, cta_shape_n, smem_shape_k, thread_shape_m, thread_shape_n)                                  \
  MATMUL_SIGNATURE(launch_##name##_##num_threads##t_cta##cta_shape_m##x##cta_shape_n##_smem##smem_shape_k##_thread##thread_shape_m##x##thread_shape_n) { \
    dim3 threads(num_threads);                                                                                                                           \
    dim3 blocks(ceil_div<int64_t>(m, cta_shape_m), ceil_div<int64_t>(n, cta_shape_n));                                                                   \
    name<num_threads, cta_shape_m, cta_shape_n, smem_shape_k, thread_shape_m, thread_shape_n><<<                                                         \
        blocks, threads, 0, nullptr>>>(m, n, k, a, lda, b, ldb, c, ldc);                                                                                 \
    CUDA_CHECK(hipGetLastError());                                                                                                                      \
  }

MATMUL_KERNEL_LAUNCH(matmul_smem_and_register_pipelining_2, 256, 128, 128, 4, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_smem_and_register_pipelining_2, 256, 128, 128, 8, 8, 8);
// MATMUL_KERNEL_LAUNCH(matmul_smem_and_register_pipelining_2, 256, 128, 128, 16, 8, 8);
// MATMUL_KERNEL_LAUNCH(matmul_smem_and_register_pipelining_2, 256, 128, 128, 24, 8, 8);
// MATMUL_KERNEL_LAUNCH(matmul_smem_and_register_pipelining_2, 256, 128, 128, 32, 8, 8);
// MATMUL_KERNEL_LAUNCH(matmul_smem_and_register_pipelining_2, 256, 128, 128, 40, 8, 8);
// MATMUL_KERNEL_LAUNCH(matmul_smem_and_register_pipelining_2, 256, 128, 128, 48, 8, 8);

MATMUL_DMODULE(m) {
  REGISTER(launch_matmul_smem_and_register_pipelining_2_256t_cta128x128_smem4_thread8x8);
  REGISTER(launch_matmul_smem_and_register_pipelining_2_256t_cta128x128_smem8_thread8x8);
  // REGISTER(launch_matmul_smem_and_register_pipelining_2_256t_cta128x128_smem16_thread8x8);
  // REGISTER(launch_matmul_smem_and_register_pipelining_2_256t_cta128x128_smem24_thread8x8);
  // REGISTER(launch_matmul_smem_and_register_pipelining_2_256t_cta128x128_smem32_thread8x8);
  // REGISTER(launch_matmul_smem_and_register_pipelining_2_256t_cta128x128_smem40_thread8x8);
  // REGISTER(launch_matmul_smem_and_register_pipelining_2_256t_cta128x128_smem48_thread8x8);
}

}  // namespace column_major
