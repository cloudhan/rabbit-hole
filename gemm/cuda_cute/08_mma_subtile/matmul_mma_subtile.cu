#include "hip/hip_runtime.h"
#include <cute/layout.hpp>                     // make_shape, make_stride, make_layout
#include <cute/pointer.hpp>                    // make_gmem_ptr
#include <cute/tensor.hpp>                     // make_tensor
#include <cute/numeric/integral_constant.hpp>  // _1

#include "cuda/matmul.cuh"

using namespace cute;

// not going to implement copy/store with multiple batches in this kernel.
template <int NumThreads, int SmemShapeM, int SmemShapeK, int VecSize, typename RegT, typename GmemT, typename CoordT>
__device__ void load_global_a(RegT& rA, const GmemT& gA, const CoordT& cA, int m, int k) {
  static_assert(is_rmem<typename RegT::engine_type>());
  static_assert(VecSize == 1 || VecSize == 2 || VecSize == 4);
  constexpr const auto ThrVal = make_layout(make_layout(Int<NumThreads>{}, Int<VecSize>{}), make_layout(Int<VecSize>{}));
  const auto ld_gA = gA.compose(ThrVal)(threadIdx.x, _);
  const auto ld_cA = cA.compose(ThrVal);
  clear(rA);
#pragma unroll
  for (int i = 0; i < VecSize; i++) {
    if (elem_less(ld_cA(threadIdx.x, i), make_coord(m, k))) {
      rA(i) = ld_gA(i);
    }
  }
}

template <int NumThreads, int SmemShapeN, int SmemShapeK, int VecSize, typename RegT, typename GmemT, typename CoordT>
__device__ void load_global_b(RegT& rB, const GmemT& gB, const CoordT& cB, int n, int k) {
  static_assert(is_rmem<typename RegT::engine_type>());
  static_assert(VecSize == 1 || VecSize == 2 || VecSize == 4);
  constexpr const auto ThrVal = make_layout(
      make_layout(make_shape(Int<SmemShapeK>{}, Int<SmemShapeN / VecSize>{}), make_stride(Int<SmemShapeN>{}, Int<VecSize>{})),
      make_layout(make_shape(Int<VecSize>{}), make_stride(Int<1>{}))
  );
  const auto ld_gB = gB.compose(ThrVal)(threadIdx.x, _);
  const auto ld_cB = cB.compose(ThrVal);
  clear(rB);
#pragma unroll
  for (int i = 0; i < VecSize; i++) {
    if (elem_less(ld_cB(threadIdx.x, i), make_coord(n, k))) {
      rB(i) = ld_gB(i);
    }
  }
}

template <int NumThreads, int SmemShapeM, int SmemShapeK, int VecSize, typename SmemT, typename RegT>
__device__ void store_smem_a(SmemT&& sA, const RegT& rA, int m, int k) {
  static_assert(is_rmem<typename RegT::engine_type>());
  static_assert(VecSize == 1 || VecSize == 2 || VecSize == 4);
  constexpr const auto ThrVal = make_layout(make_layout(Int<NumThreads>{}, Int<VecSize>{}), make_layout(Int<VecSize>{}));
  auto st_sA = sA.compose(ThrVal)(threadIdx.x, _);
  copy(rA, st_sA);
}

template <int NumThreads, int SmemShapeN, int SmemShapeK, int VecSize, typename SmemT, typename RegT>
__device__ void store_smem_b(SmemT&& sB, const RegT& rB, int n, int k) {
  static_assert(is_rmem<typename RegT::engine_type>());
  static_assert(VecSize == 1 || VecSize == 2 || VecSize == 4);
  constexpr const auto ThrVal = make_layout(
      make_layout(make_shape(Int<SmemShapeK>{}, Int<SmemShapeN / VecSize>{}), make_stride(Int<SmemShapeN>{}, Int<VecSize>{})),
      make_layout(make_shape(Int<VecSize>{}), make_stride(Int<1>{}))
  );
  auto st_sB = sB.compose(ThrVal)(threadIdx.x, _);
  copy(rB, st_sB);
}

template <typename AccT, typename FragAT, typename FragBT>
__device__ void rank1_update(const FragAT& fragA, const FragBT& fragB, AccT& acc) {
  static_assert(is_rmem<typename AccT::engine_type>() && is_rmem<typename FragAT::engine_type>() && is_rmem<typename FragBT::engine_type>());
#pragma unroll
  for (int jj = 0; jj < size<1, 1>(typename AccT::layout_type{}); jj++) {
#pragma unroll
    for (int ii = 0; ii < size<0, 1>(typename AccT::layout_type{}); ii++) {
#pragma unroll
      for (int j = 0; j < size<1, 0>(typename AccT::layout_type{}); j++) {
#pragma unroll
        for (int i = 0; i < size<0, 0>(typename AccT::layout_type{}); i++) {
          acc(make_coord(i, ii), make_coord(j, jj)) += fragA(i, ii) * fragB(j, jj);
        }
      }
    }
  }
}

template <int SubTileStepM, int SubTileStepN, typename AccT, typename CtaCT, typename CoordT>
__device__ void acc_store(const AccT& acc, CtaCT& threadC, CoordT thread_cC, int m, int n) {
  if (elem_less(thread_cC(size<0>(thread_cC) - 1, size<1>(thread_cC) - 1), make_coord(m, n))) {  // fast path
    copy(acc, threadC);
  } else {
    auto predA = make_tensor<bool>(get<0>(thread_cC.shape()));
    for (int i = 0; i < size(predA); i++) {
      predA(i) = get<0>(thread_cC(i, 0)) < m;
    }
#pragma unroll
    for (int j = 0; j < size<1>(typename AccT::layout_type{}); j++) {
      if (get<0>(thread_cC(0, j)) < n) {
        copy_if(predA, acc(_, j), threadC(_, j));
      }
    }
  }
}

__forceinline__ __device__ auto lane_id() {
  uint32_t laneid;
  asm("mov.u32 %0, %%laneid;" : "=r"(laneid) :);
  return laneid;
}

__forceinline__ __device__ auto warp_id() {
  uint32_t warpid;
  asm("mov.u32 %0, %%warpid;" : "=r"(warpid) :);
  return warpid;
}

namespace column_major {
template <int NumThreads, int CtaShapeM, int CtaShapeN, int SmemShapeK, int WarpShapeM, int WarpShapeN, int ThreadShapeM, int ThreadShapeN>
__launch_bounds__(NumThreads, 2)
    MATMUL_KERNEL_SIGNATURE(matmul_kernel_mma_subtile) {
  constexpr const auto SmemShapeM = CtaShapeM;
  constexpr const auto SmemShapeN = CtaShapeN;

  // original matrix
  const auto mA = make_tensor(make_gmem_ptr(a), make_layout(make_shape(m, k), make_stride(_1{}, lda)));  // col-major, indexed as (m, k)
  const auto mB = make_tensor(make_gmem_ptr(b), make_layout(make_shape(n, k), make_stride(ldb, _1{})));  // col-major storage, row-major indexing, indexed as (n, k), effectively a "transposed view"
  auto mC = make_tensor(make_gmem_ptr(c), make_layout(make_shape(m, n), make_stride(_1{}, ldc)));        // col-major, indexed as (m, n)

  // coordinate matrix
  const auto cA = make_identity_tensor(make_shape(m, k));
  const auto cB = make_identity_tensor(make_shape(n, k));
  const auto cC = make_identity_tensor(make_shape(m, n));

  const auto CtaShape = make_shape(Int<SmemShapeM>{}, Int<SmemShapeN>{}, Int<SmemShapeK>{});
  const auto cta_coord = make_coord(blockIdx.x, blockIdx.y, _);

  // a local view (in CuTe term, local tile) this CTA will need to process
  const auto ctaA = local_tile(mA, CtaShape, cta_coord, make_step(_1{}, _, _1{}));
  const auto ctaB = local_tile(mB, CtaShape, cta_coord, make_step(_, _1{}, _1{}));
  auto ctaC = local_tile(mC, CtaShape, cta_coord, make_step(_1{}, _1{}, _));

  const auto cta_cA = local_tile(cA, CtaShape, cta_coord, make_step(_1{}, _, _1{}));
  const auto cta_cB = local_tile(cB, CtaShape, cta_coord, make_step(_, _1{}, _1{}));
  const auto cta_cC = local_tile(cC, CtaShape, cta_coord, make_step(_1{}, _1{}, _));

  constexpr const int SmemAPad = 0;
  constexpr const int SmemBPad = 4;
  constexpr const auto SmemALayout = make_layout(make_shape(Int<SmemShapeM>{}, Int<SmemShapeK>{}), make_stride(_1{}, Int<SmemShapeM + SmemAPad>{}));
  constexpr const auto SmemBLayout = make_layout(make_shape(Int<SmemShapeN>{}, Int<SmemShapeK>{}), make_stride(_1{}, Int<SmemShapeN + SmemBPad>{}));
  // double buffering
  __shared__ float smemA[cosize(SmemALayout)];
  __shared__ float smemB[cosize(SmemBLayout)];
  auto sA = make_tensor(make_smem_ptr(smemA), SmemALayout);  // (m, k)
  auto sB = make_tensor(make_smem_ptr(smemB), SmemBLayout);  // (n, k)

  const auto stripe_gA = local_tile(ctaA, make_tile(Int<SmemShapeM>{}, Int<SmemShapeK>{}), make_coord(blockIdx.x, _));  // A(blockIdx.x*SmemShapeM:(blockIdx.x+1)*SmemShapeM, :)
  const auto stripe_gB = local_tile(ctaB, make_tile(Int<SmemShapeN>{}, Int<SmemShapeK>{}), make_coord(blockIdx.y, _));  // B(blockIdx.y*SmemShapeN:(blockIdx.y+1)*SmemShapeN, :)

  const auto stripe_cA = local_tile(cta_cA, make_tile(Int<SmemShapeM>{}, Int<SmemShapeK>{}), make_coord(blockIdx.x, _));
  const auto stripe_cB = local_tile(cta_cB, make_tile(Int<SmemShapeN>{}, Int<SmemShapeK>{}), make_coord(blockIdx.y, _));

  auto fragA = make_fragment_like<float>(make_shape(Int<ThreadShapeM / 2>{}, _2{}, _2{}));                                                 // (frag_idx, sub_tile_idx, buffering_idx)
  auto fragB = make_fragment_like<float>(make_shape(Int<ThreadShapeN / 2>{}, _2{}, _2{}));                                                 // (frag_idx, sub_tile_idx, buffering_idx)
  auto acc = make_fragment_like<float>(make_shape(make_shape(Int<ThreadShapeM / 2>{}, _2{}), make_shape(Int<ThreadShapeN / 2>{}, _2{})));  // ((i, ii), (j, jj))
  clear(acc);

  constexpr const auto SmemALoadStoreVec = (SmemShapeM * SmemShapeK) / NumThreads;  // VecSize
  constexpr const auto SmemBLoadStoreVec = (SmemShapeN * SmemShapeK) / NumThreads;  // VecSize
  auto staging_a = make_fragment_like<float>(Int<SmemALoadStoreVec>{});
  auto staging_b = make_fragment_like<float>(Int<SmemBLoadStoreVec>{});

  int p_tile_count = (k - 1) / SmemShapeK + 1;
  int p_tile_next = 0;

  load_global_a<NumThreads, SmemShapeM, SmemShapeK, SmemALoadStoreVec>(staging_a, stripe_gA(_, _, _0{}, p_tile_next), stripe_cA(_, _, _0{}, p_tile_next), m, k);
  load_global_b<NumThreads, SmemShapeN, SmemShapeK, SmemBLoadStoreVec>(staging_b, stripe_gB(_, _, _0{}, p_tile_next), stripe_cB(_, _, _0{}, p_tile_next), n, k);
  if (--p_tile_count > 0) {
    ++p_tile_next;
  }
  store_smem_a<NumThreads, SmemShapeM, SmemShapeK, SmemALoadStoreVec>(sA, staging_a, m, k);
  store_smem_b<NumThreads, SmemShapeN, SmemShapeK, SmemBLoadStoreVec>(sB, staging_b, n, k);
  __syncthreads();

  constexpr const auto CtaLayout = make_layout(make_shape(Int<CtaShapeM / ThreadShapeM>{}, Int<CtaShapeN / ThreadShapeN>{}));
  constexpr const auto WarpTile = make_tile(Int<WarpShapeM / ThreadShapeM>{}, Int<WarpShapeN / ThreadShapeN>{});
  constexpr const auto LaneWarp = zipped_divide(CtaLayout, WarpTile);  // ((lane),(warp)):(...), map from threadIdx.x to warp tiled index
  // constexpr const auto MN = logical_divide(CtaLayout, WarpTile);       // ((CtaM),(CtaN)):(...)
  // const auto remapped_coord = CtaLayout[LaneWarp(lane_id(), warp_id())];  // map to then unmap from warp tiled index, to get i,j coord.

  const auto remapped_coord = CtaLayout[LaneWarp(threadIdx.x)];     // NOTE: it is weird that LaneWarp(threadIdx.x) is faster than LaneWarp(lane_id(), warp_id())
  const auto cta_thread_i = get<0>(remapped_coord) * ThreadShapeM;  // Again, scale by ThreadShapeM not necessary,
  const auto cta_thread_j = get<1>(remapped_coord) * ThreadShapeN;  // just to keep the semantics identical with cuda impls
  constexpr const auto SubTileStepM = WarpShapeM / 2;
  constexpr const auto SubTileStepN = WarpShapeN / 2;
  // This time, for ThreadShape 8x8, we slice 2x2 (with stride in between) tiles of 4x4 of data tile. This removes all bank conflicts.
  const auto stripe_sA = local_tile(sA, make_tile(make_layout(make_shape(Int<ThreadShapeM / 2>{}, _2{}), make_stride(_1{}, Int<SubTileStepM>{})), Int<SmemShapeK>{}), make_coord(cta_thread_i / ThreadShapeM));
  const auto stripe_sB = local_tile(sB, make_tile(make_layout(make_shape(Int<ThreadShapeN / 2>{}, _2{}), make_stride(_1{}, Int<SubTileStepN>{})), Int<SmemShapeK>{}), make_coord(cta_thread_j / ThreadShapeN));

  copy(stripe_sA(_, 0, _0{}), fragA(_, _, 0));  // load_fragment a
  copy(stripe_sB(_, 0, _0{}), fragB(_, _, 0));  // load_fragment b

  const auto num_smem_block = size<3>(stripe_gA);
#pragma unroll 1  // no unroll
  for (int block_p = 0; block_p < num_smem_block; block_p++) {
#pragma unroll
    for (int smem_AB_thread_p = 0; smem_AB_thread_p < SmemShapeK; smem_AB_thread_p++) {
      if (smem_AB_thread_p == SmemShapeK - 1) {
        __syncthreads();
        store_smem_a<NumThreads, SmemShapeM, SmemShapeK, SmemALoadStoreVec>(sA, staging_a, m, k);
        store_smem_b<NumThreads, SmemShapeN, SmemShapeK, SmemBLoadStoreVec>(sB, staging_b, n, k);
        __syncthreads();
        if (--p_tile_count > 0) {
          ++p_tile_next;
        }
      }

      copy(stripe_sA(_, (smem_AB_thread_p + 1) % SmemShapeK, _0{}), fragA(_, _, (smem_AB_thread_p + 1) % 2));  // load_fragment a
      copy(stripe_sB(_, (smem_AB_thread_p + 1) % SmemShapeK, _0{}), fragB(_, _, (smem_AB_thread_p + 1) % 2));  // load_fragment b

      if (smem_AB_thread_p == 0) {
        load_global_a<NumThreads, SmemShapeM, SmemShapeK, SmemALoadStoreVec>(staging_a, stripe_gA(_, _, _0{}, p_tile_next), stripe_cA(_, _, _0{}, p_tile_next), m, k);
        load_global_b<NumThreads, SmemShapeN, SmemShapeK, SmemBLoadStoreVec>(staging_b, stripe_gB(_, _, _0{}, p_tile_next), stripe_cB(_, _, _0{}, p_tile_next), n, k);
      }

      rank1_update(fragA(_, _, smem_AB_thread_p % 2), fragB(_, _, smem_AB_thread_p % 2), acc);
    }
  }

  constexpr const auto Tiler = make_tile(
      make_layout(make_shape(Int<ThreadShapeM / 2>{}, _2{}), make_stride(_1{}, Int<SubTileStepM>{})),
      make_layout(make_shape(Int<ThreadShapeN / 2>{}, _2{}), make_stride(_1{}, Int<SubTileStepN>{}))
  );
  auto threadC = local_tile(ctaC, Tiler, make_coord(cta_thread_i / ThreadShapeM, cta_thread_j / ThreadShapeN));
  auto thread_cC = local_tile(cta_cC, Tiler, make_coord(cta_thread_i / ThreadShapeM, cta_thread_j / ThreadShapeN));
  acc_store<SubTileStepM, SubTileStepN>(acc, threadC, thread_cC, m, n);
}

#define MATMUL_KERNEL_LAUNCH(name, num_threads, cta_shape_m, cta_shape_n, smem_shape_k, warp_shape_m, warp_shape_n, thread_shape_m, thread_shape_n)                                            \
  MATMUL_SIGNATURE(launch_##name##_##num_threads##t_cta##cta_shape_m##x##cta_shape_n##_smem##smem_shape_k##_warp##warp_shape_m##x##warp_shape_n##_thread##thread_shape_m##x##thread_shape_n) { \
    dim3 threads(num_threads);                                                                                                                                                                 \
    dim3 blocks(ceil_div<int64_t>(m, cta_shape_m), ceil_div<int64_t>(n, cta_shape_n));                                                                                                         \
    name<num_threads, cta_shape_m, cta_shape_n, smem_shape_k, warp_shape_m, warp_shape_n, thread_shape_m, thread_shape_n><<<                                                                   \
        blocks, threads, 0, nullptr>>>(m, n, k, a, lda, b, ldb, c, ldc);                                                                                                                       \
    CUDA_CHECK(hipGetLastError());                                                                                                                                                            \
  }

// MATMUL_KERNEL_LAUNCH(matmul_kernel_mma_subtile, 256, 128, 128, 4, 32, 64, 8, 8);
// MATMUL_KERNEL_LAUNCH(matmul_kernel_mma_subtile, 256, 128, 128, 4, 64, 32, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_mma_subtile, 256, 128, 128, 8, 32, 64, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_mma_subtile, 256, 128, 128, 8, 64, 32, 8, 8);

MATMUL_DMODULE(m) {
  // REGISTER(launch_matmul_kernel_mma_subtile_256t_cta128x128_smem4_warp32x64_thread8x8);
  // REGISTER(launch_matmul_kernel_mma_subtile_256t_cta128x128_smem4_warp64x32_thread8x8);
  REGISTER(launch_matmul_kernel_mma_subtile_256t_cta128x128_smem8_warp32x64_thread8x8);
  REGISTER(launch_matmul_kernel_mma_subtile_256t_cta128x128_smem8_warp64x32_thread8x8);
}

}  // namespace column_major
