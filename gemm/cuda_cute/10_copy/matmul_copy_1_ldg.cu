#include "hip/hip_runtime.h"
#include <cute/layout.hpp>                     // make_shape, make_stride, make_layout
#include <cute/pointer.hpp>                    // make_gmem_ptr
#include <cute/tensor.hpp>                     // make_tensor
#include <cute/numeric/integral_constant.hpp>  // _1

#include "cuda/matmul.cuh"

using namespace cute;

// copy/store with multiple batches in this kernel is automatically achieved via TiledCopy
template <typename TiledCopy, typename ThrCopy, typename GmemT, typename CoordT, typename BoundT, typename RegT>
__device__ void ldg(
    const TiledCopy& tiled_copy, const ThrCopy& thr_copy,
    const GmemT& tensor, const CoordT& coord, const BoundT& coord_bound, RegT& reg
) {
  static_assert(is_gmem<typename GmemT::engine_type>());
  static_assert(is_rmem<typename RegT::engine_type>());
  auto tv = thr_copy.partition_S(tensor);
  auto tc = thr_copy.partition_S(coord);
  if (elem_less(tc(size(tv)), coord_bound)) {
    // https://github.com/NVIDIA/cutlass/issues/1272
    copy_vec<float>(tv, reg);
  } else {
#pragma unroll
    for (int i = 0; i < size(tv); i++) {
      reg(i) = elem_less(tc(i), coord_bound) ? tv(i) : 0;
    }
  }
}

template <int SubTileStepM, int SubTileStepN, typename AccT, typename CtaCT, typename CoordT>
__device__ void acc_store(const AccT& acc, CtaCT& threadC, CoordT thread_cC, int m, int n) {
  if (elem_less(thread_cC(size<0>(thread_cC) - 1, size<1>(thread_cC) - 1), make_coord(m, n))) {  // fast path
    copy(acc, threadC);
  } else {
    auto predA = make_tensor<bool>(get<0>(thread_cC.shape()));
    for (int i = 0; i < size(predA); i++) {
      predA(i) = get<0>(thread_cC(i, 0)) < m;
    }
#pragma unroll
    for (int j = 0; j < size<1>(typename AccT::layout_type{}); j++) {
      if (get<0>(thread_cC(0, j)) < n) {
        copy_if(predA, acc(_, j), threadC(_, j));
      }
    }
  }
}

__forceinline__ __device__ auto lane_id() {
  uint32_t laneid;
  asm("mov.u32 %0, %%laneid;" : "=r"(laneid) :);
  return laneid;
}

__forceinline__ __device__ auto warp_id() {
  uint32_t warpid;
  asm("mov.u32 %0, %%warpid;" : "=r"(warpid) :);
  return warpid;
}

namespace column_major {
template <int NumThreads, int CtaShapeM, int CtaShapeN, int SmemShapeK, int WarpShapeM, int WarpShapeN, int ThreadShapeM, int ThreadShapeN>
__launch_bounds__(NumThreads, 2)
    MATMUL_KERNEL_SIGNATURE(matmul_kernel_copy_1_ldg) {
  constexpr const auto SmemShapeM = CtaShapeM;
  constexpr const auto SmemShapeN = CtaShapeN;

  // original matrix
  const auto mA = make_tensor(make_gmem_ptr(a), make_layout(make_shape(m, k), make_stride(_1{}, lda)));  // col-major, indexed as (m, k)
  const auto mB = make_tensor(make_gmem_ptr(b), make_layout(make_shape(n, k), make_stride(ldb, _1{})));  // col-major storage, row-major indexing, indexed as (n, k), effectively a "transposed view"
  auto mC = make_tensor(make_gmem_ptr(c), make_layout(make_shape(m, n), make_stride(_1{}, ldc)));        // col-major, indexed as (m, n)

  // coordinate matrix
  const auto cA = make_identity_tensor(make_shape(m, k));
  const auto cB = make_identity_tensor(make_shape(n, k));
  const auto cC = make_identity_tensor(make_shape(m, n));

  const auto CtaShape = make_shape(Int<SmemShapeM>{}, Int<SmemShapeN>{}, Int<SmemShapeK>{});
  const auto cta_coord = make_coord(blockIdx.x, blockIdx.y, _);

  // a local view (in CuTe term, local tile) this CTA will need to process
  const auto ctaA = local_tile(mA, CtaShape, cta_coord, make_step(_1{}, _, _1{}));
  const auto ctaB = local_tile(mB, CtaShape, cta_coord, make_step(_, _1{}, _1{}));
  auto ctaC = local_tile(mC, CtaShape, cta_coord, make_step(_1{}, _1{}, _));

  const auto cta_cA = local_tile(cA, CtaShape, cta_coord, make_step(_1{}, _, _1{}));
  const auto cta_cB = local_tile(cB, CtaShape, cta_coord, make_step(_, _1{}, _1{}));
  const auto cta_cC = local_tile(cC, CtaShape, cta_coord, make_step(_1{}, _1{}, _));

  constexpr const int SmemAPad = 0;
  constexpr const int SmemBPad = 4;
  constexpr const auto SmemALayout = make_layout(make_shape(Int<SmemShapeM>{}, Int<SmemShapeK>{}), make_stride(_1{}, Int<SmemShapeM + SmemAPad>{}));
  constexpr const auto SmemBLayout = make_layout(make_shape(Int<SmemShapeN>{}, Int<SmemShapeK>{}), make_stride(_1{}, Int<SmemShapeN + SmemBPad>{}));
  // double buffering
  __shared__ float smemA[cosize(SmemALayout)];
  __shared__ float smemB[cosize(SmemBLayout)];
  auto sA = make_tensor(make_smem_ptr(smemA), SmemALayout);  // (m, k)
  auto sB = make_tensor(make_smem_ptr(smemB), SmemBLayout);  // (n, k)

  const auto stripe_gA = local_tile(ctaA, make_tile(Int<SmemShapeM>{}, Int<SmemShapeK>{}), make_coord(blockIdx.x, _));  // A(blockIdx.x*SmemShapeM:(blockIdx.x+1)*SmemShapeM, :)
  const auto stripe_gB = local_tile(ctaB, make_tile(Int<SmemShapeN>{}, Int<SmemShapeK>{}), make_coord(blockIdx.y, _));  // B(blockIdx.y*SmemShapeN:(blockIdx.y+1)*SmemShapeN, :)

  const auto stripe_cA = local_tile(cta_cA, make_tile(Int<SmemShapeM>{}, Int<SmemShapeK>{}), make_coord(blockIdx.x, _));
  const auto stripe_cB = local_tile(cta_cB, make_tile(Int<SmemShapeN>{}, Int<SmemShapeK>{}), make_coord(blockIdx.y, _));

  auto fragA = make_fragment_like<float>(make_shape(Int<ThreadShapeM / 2>{}, _2{}, _2{}));                                                 // (frag_idx, sub_tile_idx, buffering_idx)
  auto fragB = make_fragment_like<float>(make_shape(Int<ThreadShapeN / 2>{}, _2{}, _2{}));                                                 // (frag_idx, sub_tile_idx, buffering_idx)
  auto acc = make_fragment_like<float>(make_shape(make_shape(Int<ThreadShapeM / 2>{}, _2{}), make_shape(Int<ThreadShapeN / 2>{}, _2{})));  // ((i, ii), (j, jj))
  clear(acc);

  constexpr const auto SmemALoadStoreVec = std::min(4, (SmemShapeM * SmemShapeK) / NumThreads);
  constexpr const auto SmemBLoadStoreVec = std::min(4, (SmemShapeN * SmemShapeK) / NumThreads);
  constexpr const auto SmemALoadStoreBatch = (SmemShapeM * SmemShapeK) / (NumThreads * SmemALoadStoreVec);
  constexpr const auto SmemBLoadStoreBatch = (SmemShapeN * SmemShapeK) / (NumThreads * SmemBLoadStoreVec);
  static_assert(SmemShapeM % (SmemALoadStoreVec * SmemALoadStoreBatch) == 0);
  static_assert(SmemShapeN % (SmemBLoadStoreVec * SmemBLoadStoreBatch) == 0);

  const auto tiled_copy_smem_a = make_tiled_copy(
      Copy_Atom<AutoVectorizingCopyWithAssumedAlignment<32>, float>{},  // For store only
      make_layout(make_shape(Int<SmemShapeM / (SmemALoadStoreVec * SmemALoadStoreBatch)>{}, Int<SmemShapeK>{})),
      make_layout(make_shape(Int<SmemALoadStoreVec>{}))
  );
  const auto tiled_copy_smem_b = make_tiled_copy(
      Copy_Atom<AutoVectorizingCopyWithAssumedAlignment<32>, float>{},  // For store only
      make_layout(make_shape(Int<SmemShapeN / (SmemBLoadStoreVec * SmemBLoadStoreBatch)>{}, Int<SmemShapeK>{}), LayoutRight{}),
      make_layout(make_shape(Int<SmemBLoadStoreVec>{}))
  );
  auto thr_copy_smem_a = tiled_copy_smem_a.get_thread_slice(threadIdx.x);
  auto thr_copy_smem_b = tiled_copy_smem_b.get_thread_slice(threadIdx.x);
  auto thr_store_smem_a_view = thr_copy_smem_a.partition_D(sA);
  auto thr_store_smem_b_view = thr_copy_smem_b.partition_D(sB);

  auto staging_a = make_fragment_like<float>(shape(thr_store_smem_a_view));
  auto staging_b = make_fragment_like<float>(shape(thr_store_smem_b_view));

  int p_tile_count = (k - 1) / SmemShapeK + 1;
  int p_tile_next = 0;

  ldg(tiled_copy_smem_a, thr_copy_smem_a, stripe_gA(_, _, _0{}, p_tile_next), stripe_cA(_, _, _0{}, p_tile_next), shape(mA), staging_a);
  ldg(tiled_copy_smem_b, thr_copy_smem_b, stripe_gB(_, _, _0{}, p_tile_next), stripe_cB(_, _, _0{}, p_tile_next), shape(mB), staging_b);
  if (--p_tile_count > 0) {
    ++p_tile_next;
  }
  copy(tiled_copy_smem_a, staging_a, thr_store_smem_a_view);
  copy(tiled_copy_smem_b, staging_b, thr_store_smem_b_view);
  __syncthreads();

  constexpr const auto CtaLayout = make_layout(make_shape(Int<CtaShapeM / ThreadShapeM>{}, Int<CtaShapeN / ThreadShapeN>{}));
  constexpr const auto WarpTile = make_tile(Int<WarpShapeM / ThreadShapeM>{}, Int<WarpShapeN / ThreadShapeN>{});
  constexpr const auto LaneWarp = zipped_divide(CtaLayout, WarpTile);  // ((lane),(warp)):(...), map from threadIdx.x to warp tiled index
  // constexpr const auto MN = logical_divide(CtaLayout, WarpTile);       // ((CtaM),(CtaN)):(...)
  // const auto remapped_coord = CtaLayout[LaneWarp(lane_id(), warp_id())];  // map to then unmap from warp tiled index, to get i,j coord.

  const auto remapped_coord = CtaLayout[LaneWarp(threadIdx.x)];     // NOTE: it is weird that LaneWarp(threadIdx.x) is faster than LaneWarp(lane_id(), warp_id())
  const auto cta_thread_i = get<0>(remapped_coord) * ThreadShapeM;  // Again, scale by ThreadShapeM not necessary,
  const auto cta_thread_j = get<1>(remapped_coord) * ThreadShapeN;  // just to keep the semantics identical with cuda impls
  constexpr const auto SubTileStepM = WarpShapeM / 2;
  constexpr const auto SubTileStepN = WarpShapeN / 2;
  // This time, for ThreadShape 8x8, we slice 2x2 (with stride in between) tiles of 4x4 of data tile. This removes all bank conflicts.
  const auto stripe_sA = local_tile(sA, make_tile(make_layout(make_shape(Int<ThreadShapeM / 2>{}, _2{}), make_stride(_1{}, Int<SubTileStepM>{})), Int<SmemShapeK>{}), make_coord(cta_thread_i / ThreadShapeM));
  const auto stripe_sB = local_tile(sB, make_tile(make_layout(make_shape(Int<ThreadShapeN / 2>{}, _2{}), make_stride(_1{}, Int<SubTileStepN>{})), Int<SmemShapeK>{}), make_coord(cta_thread_j / ThreadShapeN));

  copy(stripe_sA(_, 0, _0{}), fragA(_, _, 0));  // load_fragment a
  copy(stripe_sB(_, 0, _0{}), fragB(_, _, 0));  // load_fragment b

  const auto num_smem_block = size<3>(stripe_gA);
#pragma unroll 1  // no unroll
  for (int block_p = 0; block_p < num_smem_block; block_p++) {
#pragma unroll
    for (int smem_AB_thread_p = 0; smem_AB_thread_p < SmemShapeK; smem_AB_thread_p++) {
      if (smem_AB_thread_p == SmemShapeK - 1) {
        __syncthreads();
        copy(tiled_copy_smem_a, staging_a, thr_store_smem_a_view);
        copy(tiled_copy_smem_b, staging_b, thr_store_smem_b_view);
        __syncthreads();
        if (--p_tile_count > 0) {
          ++p_tile_next;
        }
      }

      copy(stripe_sA(_, (smem_AB_thread_p + 1) % SmemShapeK, _0{}), fragA(_, _, (smem_AB_thread_p + 1) % 2));  // load_fragment a
      copy(stripe_sB(_, (smem_AB_thread_p + 1) % SmemShapeK, _0{}), fragB(_, _, (smem_AB_thread_p + 1) % 2));  // load_fragment b

      if (smem_AB_thread_p == 0) {
        ldg(tiled_copy_smem_a, thr_copy_smem_a, stripe_gA(_, _, _0{}, p_tile_next), stripe_cA(_, _, _0{}, p_tile_next), shape(mA), staging_a);
        ldg(tiled_copy_smem_b, thr_copy_smem_b, stripe_gB(_, _, _0{}, p_tile_next), stripe_cB(_, _, _0{}, p_tile_next), shape(mB), staging_b);
      }

      // Just a simple reorganization, it allows us to dispatch to (M) x (N) => (M,N) version of gemm defined
      // include/cute/algorithm/gemm.hpp
      auto fa = group_modes<0, 2>(fragA(_, _, smem_AB_thread_p % 2));  // ((ThreadShapeM/2,2)):...
      auto fb = group_modes<0, 2>(fragB(_, _, smem_AB_thread_p % 2));  // ((ThreadShapeN/2,2)):...
      gemm(fa, fb, acc);
    }
  }

  constexpr const auto Tiler = make_tile(
      make_layout(make_shape(Int<ThreadShapeM / 2>{}, _2{}), make_stride(_1{}, Int<SubTileStepM>{})),
      make_layout(make_shape(Int<ThreadShapeN / 2>{}, _2{}), make_stride(_1{}, Int<SubTileStepN>{}))
  );
  auto threadC = local_tile(ctaC, Tiler, make_coord(cta_thread_i / ThreadShapeM, cta_thread_j / ThreadShapeN));
  auto thread_cC = local_tile(cta_cC, Tiler, make_coord(cta_thread_i / ThreadShapeM, cta_thread_j / ThreadShapeN));
  acc_store<SubTileStepM, SubTileStepN>(acc, threadC, thread_cC, m, n);
}

#define MATMUL_KERNEL_LAUNCH(name, num_threads, cta_shape_m, cta_shape_n, smem_shape_k, warp_shape_m, warp_shape_n, thread_shape_m, thread_shape_n)                                            \
  MATMUL_SIGNATURE(launch_##name##_##num_threads##t_cta##cta_shape_m##x##cta_shape_n##_smem##smem_shape_k##_warp##warp_shape_m##x##warp_shape_n##_thread##thread_shape_m##x##thread_shape_n) { \
    dim3 threads(num_threads);                                                                                                                                                                 \
    dim3 blocks(ceil_div<int64_t>(m, cta_shape_m), ceil_div<int64_t>(n, cta_shape_n));                                                                                                         \
    name<num_threads, cta_shape_m, cta_shape_n, smem_shape_k, warp_shape_m, warp_shape_n, thread_shape_m, thread_shape_n><<<                                                                   \
        blocks, threads, 0, nullptr>>>(m, n, k, a, lda, b, ldb, c, ldc);                                                                                                                       \
    CUDA_CHECK(hipGetLastError());                                                                                                                                                            \
  }

// Iter mode from TiledCopy allows us to finish a smem copy in multiple batches. Then SmemShapeK == 16 is possible now.
// But SmemShapeK == 24 is not possible, we are vectorizing and batching over M (or N for b), but SmemShapeM % (VecSize * Batch) != 0
// Vectorizing over M and batching over K is possible, but is not performant.
MATMUL_KERNEL_LAUNCH(matmul_kernel_copy_1_ldg, 256, 128, 128, 8, 32, 64, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_copy_1_ldg, 256, 128, 128, 8, 64, 32, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_copy_1_ldg, 256, 128, 128, 16, 32, 64, 8, 8);
MATMUL_KERNEL_LAUNCH(matmul_kernel_copy_1_ldg, 256, 128, 128, 16, 64, 32, 8, 8);

MATMUL_DMODULE(m) {
  REGISTER(launch_matmul_kernel_copy_1_ldg_256t_cta128x128_smem8_warp32x64_thread8x8);
  REGISTER(launch_matmul_kernel_copy_1_ldg_256t_cta128x128_smem8_warp64x32_thread8x8);
  REGISTER(launch_matmul_kernel_copy_1_ldg_256t_cta128x128_smem16_warp32x64_thread8x8);
  REGISTER(launch_matmul_kernel_copy_1_ldg_256t_cta128x128_smem16_warp64x32_thread8x8);
}

}  // namespace column_major
